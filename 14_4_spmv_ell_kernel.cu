#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include "utils.hpp"


struct ELLMatrix {
    unsigned int numRows;
    unsigned int * nnzPerRow;
    unsigned int * colIdx;
    float * value;
};

__global__ void spmv_ell_kernel(ELLMatrix ellMatrix, float *x, float *y) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < ellMatrix.numRows) {
        float sum = 0.0f;
        for (unsigned int t = 0; t < ellMatrix.nnzPerRow[row]; ++t) {
            unsigned int i = t * ellMatrix.numRows + row;
            unsigned int col = ellMatrix.colIdx[i];
            float value = ellMatrix.value[i];
            sum += x[col] * value;
        }
        y[row] = sum;
    }
}

int main() {
    unsigned int numRows = 4;
    unsigned int nnzPerRow[] = {2, 3, 2, 1};
    unsigned int colIdx[] = {0, 0, 1, 3, 1, 2, 2, 999, 999, 3, 999, 999};
    float h_value[] = {1, 5, 2, 6, 7, 3, 8, 0, 0, 9, 0, 0}; // Non-zero values
    float h_x[] = {1.0f, 2.0f, 3.0f, 4.0f}; // Input vector
    float h_y[] = {0.0f, 0.0f, 0.0f, 0.0f}; // Output vector (initialized to zero)

    // Device-site memory allocation
    unsigned int *d_nnzPerRow, *d_colIdx;
    float *d_value, *d_x, *d_y;
    hipMalloc(&d_nnzPerRow, sizeof(nnzPerRow));
    hipMalloc(&d_colIdx, sizeof(colIdx));
    hipMalloc(&d_value, sizeof (h_value));
    hipMalloc(&d_x, sizeof(h_x));
    hipMalloc(&d_y, sizeof(h_y));

    hipMemcpy(d_nnzPerRow, nnzPerRow, sizeof(nnzPerRow), hipMemcpyHostToDevice);
    hipMemcpy(d_colIdx, colIdx, sizeof(colIdx), hipMemcpyHostToDevice);
    hipMemcpy(d_value, h_value, sizeof(h_value), hipMemcpyHostToDevice);  
    hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice);  
    hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice);

    ELLMatrix d_ellMatrix = {numRows, d_nnzPerRow, d_colIdx, d_value};

    dim3 blockDim(32);  
    dim3 gridDim((numRows + blockDim.x - 1) / blockDim.x);

    spmv_ell_kernel<<<gridDim, blockDim>>>(d_ellMatrix, d_x, d_y);

    hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost);

    float expected_y[] = {15.0f, 50.0f, 28.0f, 24.0f}; 
    for (unsigned int i = 0; i < numRows; ++i) {  
        if (fabs(h_y[i] - expected_y[i]) > 1e-3) {
            printf("%f %f\n", h_y[i], expected_y[i]);
            std::cerr << "Test failed!\n";
            // Free device memory
            hipFree(d_nnzPerRow);  
            hipFree(d_colIdx);  
            hipFree(d_value);  
            hipFree(d_x);  
            hipFree(d_y);
            return ;
        }
    }  
    std::cout << "Test passed!" << std::endl;

    hipFree(d_nnzPerRow);  
    hipFree(d_colIdx);  
    hipFree(d_value);  
    hipFree(d_x);  
    hipFree(d_y);

    return 0;  
}